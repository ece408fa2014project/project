#ifndef __FACE_DETECTION_MAIN
#define __FACE_DETECTION_MAIN
#include "../easypng/png.h"
#include "../util/png2arrays.h"
#include "kernels.cu"

void do_face_detection_cuda(float * r, float * g, float * b, int width, int height) {

    int size = width * height;
    hipError_t cuda_ret;
    //allocate the device input arrays
    float *r_dev;
    float *g_dev;
    float *b_dev;

    hipMalloc((void**) &r_dev, size * sizeof(float));
    hipMalloc((void**) &g_dev, size * sizeof(float));
    hipMalloc((void**) &b_dev, size * sizeof(float));

    //allocate device output arrays
    float *grayscale_dev_1;

    hipMalloc((void**) &grayscale_dev_1, size * sizeof(float));

    float *grayscale_dev_2;

    hipMalloc((void**) &grayscale_dev_2, size * sizeof(float));

    float *grayscale_dev_1_out;

    hipMalloc((void**) &grayscale_dev_1_out, size * sizeof(float));

    float *grayscale_dev_2_out;

    hipMalloc((void**) &grayscale_dev_2_out, size * sizeof(float));

    float *grayscale_dev_3_out;

    hipMalloc((void**) &grayscale_dev_3_out, size * sizeof(float));

    float *grayscale_dev_4_out;

    hipMalloc((void**) &grayscale_dev_4_out, size * sizeof(float));


    //copy input to device
    hipMemcpy(r_dev, r, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_dev, g, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size * sizeof(float), hipMemcpyHostToDevice);

    dim3 dim_grid_gray(width / 16 + 1, height / 16 + 1, 1);
    dim3 dim_block_gray(16, 16, 1);

    skin_detection<<<dim_grid_gray, dim_block_gray>>>(r_dev, g_dev, b_dev, grayscale_dev_1, grayscale_dev_2, width, height);

    clean_up<<<dim_grid_gray, dim_block_gray>>>(grayscale_dev_1, grayscale_dev_2, grayscale_dev_3_out, grayscale_dev_4_out, width, height);

    quantization<<<dim_grid_gray, dim_block_gray>>>(grayscale_dev_3_out, grayscale_dev_4_out, grayscale_dev_1_out, grayscale_dev_2_out, width, height);

    hipFree(r_dev);
    hipFree(g_dev);
    hipFree(b_dev);

    float * grayscale_host_skin = (float *)malloc(size * sizeof(float));
    hipMemcpy(grayscale_host_skin, grayscale_dev_1, size * sizeof(float), hipMemcpyDeviceToHost);
    float * grayscale_host_hair = (float *)malloc(size * sizeof(float));
    hipMemcpy(grayscale_host_hair, grayscale_dev_2, size * sizeof(float), hipMemcpyDeviceToHost);

    face_detection(grayscale_host_skin, grayscale_host_hair, r, g, b, width, height);

    // hipMemcpy(r, grayscale_dev_1, size * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy(g, grayscale_dev_1, size * sizeof(float), hipMemcpyDeviceToHost);
    // hipMemcpy(b, grayscale_dev_1, size * sizeof(float), hipMemcpyDeviceToHost);



    hipFree(grayscale_dev_1);
    hipFree(grayscale_dev_2);
    hipFree(grayscale_dev_1_out);
    hipFree(grayscale_dev_2_out);
    hipFree(grayscale_dev_3_out);
    hipFree(grayscale_dev_4_out);
    free(grayscale_host_skin);
    free(grayscale_host_hair);
}

#endif //__CUDA_MAIN
