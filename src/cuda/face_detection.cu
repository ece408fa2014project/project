#ifndef __FACE_DETECTION_MAIN
#define __FACE_DETECTION_MAIN
#include "../easypng/png.h"
#include "../util/png2arrays.h"
#include "kernels.cu"

void do_face_detection_cuda(float * r, float * g, float * b, float * out_r, float * out_g,  float * out_b, int width, int height) {
    
    int size = width * height;
    //allocate the device input arrays
    float *r_dev;
    float *g_dev;
    float *b_dev;
    
    hipMalloc((void**) &r_dev, size * sizeof(float));
    hipMalloc((void**) &g_dev, size * sizeof(float));
    hipMalloc((void**) &b_dev, size * sizeof(float));
    
    //allocate device output arrays
    float *grayscale_dev_1;
    
    hipMalloc((void**) &grayscale_dev_1, size * sizeof(float));
    
    float *grayscale_dev_2;
    
    hipMalloc((void**) &grayscale_dev_2, size * sizeof(float));
    
    float *grayscale_dev_1_out;
    
    hipMalloc((void**) &grayscale_dev_1_out, size * sizeof(float));
    
    float *grayscale_dev_2_out;
    
    hipMalloc((void**) &grayscale_dev_2_out, size * sizeof(float));
    
    float *grayscale_dev_3_out;
    
    hipMalloc((void**) &grayscale_dev_3_out, size * sizeof(float));
    
    float *grayscale_dev_4_out;
    
    hipMalloc((void**) &grayscale_dev_4_out, size * sizeof(float));
    
    
    //copy input to device
    hipMemcpy(r_dev, r, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_dev, g, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size * sizeof(float), hipMemcpyHostToDevice);
    
    dim3 dim_grid_gray(width / 16 + 1, height / 16 + 1, 1);
    dim3 dim_block_gray(16, 16, 1);
    
    skin_detection<<<dim_block_gray, dim_grid_gray>>>(r_dev, g_dev, b_dev, grayscale_dev_1, grayscale_dev_2, width, height);
    
    clean_up<<<dim_block_gray, dim_grid_gray>>>(grayscale_dev_1, grayscale_dev_2, grayscale_dev_3_out, grayscale_dev_4_out, width, height);
    
    quantization<<<dim_block_gray, dim_grid_gray>>>(grayscale_dev_3_out, grayscale_dev_4_out, grayscale_dev_1_out, grayscale_dev_2_out, width, height);
    
    hipFree(r_dev);
    hipFree(g_dev);
    hipFree(b_dev);
    
    float * grayscale_host_skin = (float *)malloc(size * sizeof(float));
    hipMemcpy(grayscale_host_skin, grayscale_dev_1_out, size * sizeof(float), hipMemcpyDeviceToHost);
    float * grayscale_host_hair = (float *)malloc(size * sizeof(float));
    hipMemcpy(grayscale_host_hair, grayscale_dev_2_out, size * sizeof(float), hipMemcpyDeviceToHost);
    
    face_detection(grayscale_host_skin, grayscale_host_hair, out_r, out_g, out_b, width, height);
    
    hipFree(grayscale_dev_1);
    hipFree(grayscale_dev_2);
    hipFree(grayscale_dev_1_out);
    hipFree(grayscale_dev_2_out);
    hipFree(grayscale_dev_3_out);
    hipFree(grayscale_dev_4_out);
}

#endif //__CUDA_MAIN
