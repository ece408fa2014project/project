#ifndef __CUDA_MAIN
#define __CUDA_MAIN
#include "../easypng/png.h"
#include "../util/png2arrays.h"
#include "kernels.cu"

void cuda_edge_algorithm(PNG * image) {
    png2arrays converter;
    converter.parse_png(image);

    int size = image->width() * image->height();
 
    //allocate the device input arrays
    float *r_dev;
    float *g_dev;
    float *b_dev;
 
    hipMalloc((void**) &r_dev, size * sizeof(float));
    hipMalloc((void**) &g_dev, size * sizeof(float));
    hipMalloc((void**) &b_dev, size * sizeof(float));
 
    //allocate device output arrays
    float *grayscale_dev;
 
    hipMalloc((void**) &grayscale_dev, size * sizeof(float));
 
    //copy input to device
    hipMemcpy(r_dev, converter.r, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_dev, converter.g, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, converter.b, size * sizeof(float), hipMemcpyHostToDevice);

    dim3 dim_grid_gray(image->width() / 16 + 1, image->height() / 16 + 1, 1);
    dim3 dim_block_gray(16, 16, 1);
    dim3 dim_grid_gauss(image->width()/OUTPUT_TILE_SIZE + 1,image->height()/OUTPUT_TILE_SIZE + 1, 1);
    dim3 dim_block_gauss(INPUT_TILE_SIZE, INPUT_TILE_SIZE, 1);

    grayscale_kernel<<<dim_block_gray, dim_grid_gray>>>(r_dev, g_dev, b_dev, grayscale_dev, image->width(), image->height());

    hipFree(r_dev);
    hipFree(g_dev);
    hipFree(b_dev);

    float *gray_gauss_dev;
    hipMalloc((void**) &gray_gauss_dev, size * sizeof(float));
    
    //allocate device variables for gradient calculations
    float *grad_dev;
    float *grad_x_dev;
    float *grad_y_dev;

    hipMalloc((void**) &grad_dev, size * sizeof(float));
    hipMalloc((void**) &grad_x_dev, size * sizeof(float));
    hipMalloc((void**) &grad_y_dev, size * sizeof(float));

    
}

#endif //__CUDA_MAIN
