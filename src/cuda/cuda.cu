#ifndef __CUDA_MAIN
#define __CUDA_MAIN
#include "../easypng/png.h"
#include "../util/png2arrays.h"
#include "kernels.cu"

void print_image(float * cuda_handle, string filename, int width, int height) {
    float * grayscale_host = (float *)malloc(width * height * sizeof(float));

    hipMemcpy(grayscale_host, cuda_handle, width * height * sizeof(float), hipMemcpyDeviceToHost);

    png2arrays from_grayscale;

    from_grayscale.r = grayscale_host;
    from_grayscale.g = grayscale_host;
    from_grayscale.b = grayscale_host;
    from_grayscale.x_dim = width;
    from_grayscale.y_dim = height;

    PNG * grayscale = from_grayscale.from_arrays();
    grayscale->writeToFile(filename);

    delete grayscale;
    delete grayscale_host;
}

void cuda_edge_algorithm(PNG * image) {
    png2arrays converter;
    converter.parse_png(image);

    hipError_t cuda_ret;

    int size = image->width() * image->height();
 
    //allocate the device input arrays
    float *r_dev;
    float *g_dev;
    float *b_dev;
 
    hipMalloc((void**) &r_dev, size * sizeof(float));
    hipMalloc((void**) &g_dev, size * sizeof(float));
    hipMalloc((void**) &b_dev, size * sizeof(float));
 
    //allocate device output arrays
    float *grayscale_dev;
 
    hipMalloc((void**) &grayscale_dev, size * sizeof(float));
 
    //copy input to device
    hipMemcpy(r_dev, converter.r, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_dev, converter.g, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, converter.b, size * sizeof(float), hipMemcpyHostToDevice);

    dim3 dim_grid_gray(image->width() / 16 + 1, image->height() / 16 + 1, 1);
    dim3 dim_block_gray(16, 16, 1);

    dim3 dim_grid_gauss(image->width()/OUTPUT_TILE_SIZE + 1,image->height()/OUTPUT_TILE_SIZE + 1, 1);
    dim3 dim_block_gauss(INPUT_TILE_SIZE, INPUT_TILE_SIZE, 1);
    
    grayscale_kernel<<<dim_block_gray, dim_grid_gray>>>(r_dev, g_dev, b_dev, grayscale_dev, image->width(), image->height());

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) 
    {   
        printf("%s\n", hipGetErrorString(cuda_ret));
    }
    hipFree(r_dev);
    hipFree(g_dev);
    hipFree(b_dev);

    #ifdef TEST_MODE
        print_image(grayscale_dev, "grayscale.png", image->width(), image->height());
    #endif //TEST_MODE

    float *gray_gauss_dev;
    hipMalloc((void**) &gray_gauss_dev, size * sizeof(float));

    gaussian_filter_kernel<<<dim_block_gauss, dim_grid_gauss>>>(grayscale_dev, gray_gauss_dev, image->width(), image->height());

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) 
    {   
        printf("%s\n", hipGetErrorString(cuda_ret));
    }
    #ifdef TEST_MODE
        print_image(gray_gauss_dev, "gauss.png", image->width(), image->height());
    #endif //TEST_MODE

    //allocate device variables for gradient calculations
    float *grad_dev;
    float *grad_x_dev;
    float *grad_y_dev;

    hipMalloc((void**) &grad_dev, size * sizeof(float));
    hipMalloc((void**) &grad_x_dev, size * sizeof(float));
    hipMalloc((void**) &grad_y_dev, size * sizeof(float));

    dim3 dim_grid_grad(image->width()/OUTPUT_TILE_SIZE + 1,image->height()/OUTPUT_TILE_SIZE + 1, 1);
    dim3 dim_block_grad(INPUT_TILE_SIZE_GRAD, INPUT_TILE_SIZE_GRAD, 1);
   
    gradient_calc_kernel<<<dim_block_grad, dim_grid_grad>>>(gray_gauss_dev, grad_x_dev, grad_y_dev, grad_dev, image->width(), image->height());

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) 
    {   
        printf("%s\n", hipGetErrorString(cuda_ret));
    }
    #ifdef TEST_MODE
        print_image(grad_dev, "gradients.png", image->width(), image->height());
    #endif //TEST_MODE
    hipFree(gray_gauss_dev);



    int * theta_dev;

    hipMalloc((void**) &theta_dev, size * sizeof(int));
    //the directions are labeled counterclockwise from theta=0
    theta_calc_kernel<<<dim_block_gray, dim_grid_gray>>>(grad_x_dev, grad_y_dev, theta_dev, image->width(), image->height());

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) 
    {   
        printf("%s\n", hipGetErrorString(cuda_ret));
    }
    hipFree(grad_x_dev);
    hipFree(grad_y_dev);

    dim3 dim_grid_trace(image->width()/16, image->height()/16, 1);
    dim3 dim_block_trace(18, 18, 1);

}

#endif //__CUDA_MAIN
