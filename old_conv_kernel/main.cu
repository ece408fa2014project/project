/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    // Initialize host variables ----------------------------------------------

    #if TEST_MODE
    printf("\n***Running in test mode***\n"); fflush(stdout);
    #endif

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

	Matrix M_h, N_h, P_h; // M: filter, N: input image, P: output image
	Matrix N_d, P_d;
	unsigned imageHeight, imageWidth;
	hipError_t cuda_ret;

	/* Read image dimensions */
    if (argc == 1) {
        imageHeight = 600;
        imageWidth = 1000;
    } else if (argc == 2) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[1]);
    } else if (argc == 3) {
        imageHeight = atoi(argv[1]);
        imageWidth = atoi(argv[2]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./convolution          # Image is 600 x 1000"
           "\n    Usage: ./convolution <m>      # Image is m x m"
           "\n    Usage: ./convolution <m> <n>  # Image is m x n"
           "\n");
        exit(0);
    }

	/* Allocate host memory */
	M_h = allocateMatrix(FILTER_SIZE, FILTER_SIZE);
	N_h = allocateMatrix(imageHeight, imageWidth);
	P_h = allocateMatrix(imageHeight, imageWidth);
    
	/* Initialize filter and images */
	initMatrix(M_h);
	initMatrix(N_h);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Image: %u x %u\n", imageHeight, imageWidth);
    printf("    Mask: %u x %u\n", FILTER_SIZE, FILTER_SIZE);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

	N_d = allocateDeviceMatrix(imageHeight, imageWidth);
	P_d = allocateDeviceMatrix(imageHeight, imageWidth);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

	/* Copy image to device global memory */
	copyToDeviceMatrix(N_d, N_h);

	/* Copy mask to device constant memory */
    // INSERT CODE HERE
    hipMemcpyToSymbol(HIP_SYMBOL("M_c"), M_h.elements, FILTER_SIZE * FILTER_SIZE * sizeof(float), 0, hipMemcpyHostToDevice);


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    // INSERT CODE HERE
    // Use OUTPUT_TILE_SIZE and INPUT_TILE_SIZE defined in support.h
        dim3 dim_grid(imageWidth/OUTPUT_TILE_SIZE + 1,imageHeight/OUTPUT_TILE_SIZE + 1, 1);
        dim3 dim_block(INPUT_TILE_SIZE, INPUT_TILE_SIZE, 1);

	convolution<<<dim_grid, dim_block>>>(N_d, P_d);

	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) 
        {   
            printf("%s\n", hipGetErrorString(cuda_ret));
            FATAL("Unable to launch/execute kernel");
        }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    copyFromDeviceMatrix(P_h, P_d);

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    #if TEST_MODE
    printf("\nResult:\n");
    for(int row = 0; row < P_h.height; ++row) {
        for(int col = 0; col < P_h.width; ++col) {
            printf("%.2f ", P_h.elements[row*P_h.width + col]);
        }
        printf("\n");
    }
    #endif

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(M_h, N_h, P_h);

    // Free memory ------------------------------------------------------------

	freeMatrix(M_h);
	freeMatrix(N_h);
	freeMatrix(P_h);
	freeDeviceMatrix(N_d);
	freeDeviceMatrix(P_d);

	return 0;
}

